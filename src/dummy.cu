

#include <hip/hip_runtime.h>
#include <stdio.h>


extern "C" __global__ void CudaDummy_kernel(
        const float *A, 
        const float *B,
        float *C, 
        int N) 
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) C[i] = A[i] + B[i];
}

extern "C" void CudaDummy() 
{
    int N   = 100000000;

    float *A;
    float *B;
    float *C; 

    hipMallocManaged(&A, N*sizeof(float));
    hipMallocManaged(&B, N*sizeof(float));
    hipMallocManaged(&C, N*sizeof(float));


    for(int i=0;i<N;++i){
        A[i] = (float)i;
        B[i] = (float)i;
    }

    int blockSize = 256;
    int numBlocks = (N+blockSize-1)/blockSize;

    CudaDummy_kernel<<<numBlocks,blockSize>>>(A, B, C, N);
    CudaDummy_kernel<<<numBlocks,blockSize>>>(A, B, C, N);
    CudaDummy_kernel<<<numBlocks,blockSize>>>(A, B, C, N);
    CudaDummy_kernel<<<numBlocks,blockSize>>>(A, B, C, N);
    CudaDummy_kernel<<<numBlocks,blockSize>>>(A, B, C, N);
    hipDeviceSynchronize();

    for(int i=0;i<10;++i){
        printf("%i: %f + %f = %f\n", i, A[i], B[i], C[i]);
    }

    for(int i=N-10;i<N;++i){
        printf("%i: %f + %f = %f\n", i, A[i], B[i], C[i]);
    }
}
